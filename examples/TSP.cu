
#include <hip/hip_runtime.h>
#include<random>
#include<iostream>
#include<stdio.h>
#include<ctime>

int *perm;
int count = 0;

void print(int a[], int n)
{
	for (int i=0; i<n; i++)
	{
		perm[count] = a[i];
		count++;
		std::cout<<a[i]<<" ";
	}
	std::cout<<std::endl;
}

void heapPermutation(int a[], int size, int n)
{
	if (size==1)
	{
		print(a, n);
		return;
	}
	else
	{
		for (int i=0; i<size; i++)
		{
			heapPermutation(a, size-1, n);
			if (size%2==1)
				std::swap(a[0], a[size-1]);
			else
				std::swap(a[i], a[size-1]);
		}
	}
}
void matrix_randomizer(float *arr, int size)
{
	std::random_device rd;
        std::uniform_real_distribution<float> dist(0, 1);

        std::mt19937_64 prng;
        prng.seed(dist(rd));

	for (int i=0; i<size*size; i++)
	{
		arr[i] = dist(prng);
	}
}

float cpu_call(float *arr, int *perm, float *result, int size, int total_threads)
{
	auto t1 = std::clock();
	for (int id=0; id<total_threads; id++)
	{
		//permutation
		int *pos = perm + id*size;
		//traversal
		float cost = 0;
		int last, curr;
	
		for (int i=0; i<size-1; i++)
		{	
			last = pos[i];
			curr = pos[i+1];
			cost += arr[last*size + curr];
		}
		last = pos[size-1];
		curr = pos[0];
		cost += arr[last*size + curr];
		result[id] = cost;
	}
	auto t2 = std::clock();
	return static_cast<float>(t2-t1)/CLOCKS_PER_SEC;

}
__global__ void TSP(int *perm, float *result, float *arr, int size, int total_threads)
{
	int block_id = blockIdx.x;
	int thread_id = threadIdx.x;
	int threads = blockDim.x;
	int id = thread_id + block_id * threads;
	if (id >= total_threads) return;
	//permutation
	int *pos = perm + id*size;
	//traversal
	float cost = 0;
	int last, curr;
	
	for (int i=0; i<size-1; i++)
	{	
		last = pos[i];
		curr = pos[i+1];
		cost += arr[last*size + curr];
	}
	last = pos[size-1];
	curr = pos[0];
	cost += arr[last*size + curr];
	result[id] = cost;
}
float gpu_call(float *arr, int *perm, float *result, int size, int total_threads)
{
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	int MaxThreadsPerBlock = prop.maxThreadsPerBlock;
	int HmBlocks = 1 + total_threads/MaxThreadsPerBlock;

	dim3 BlocksPerGrid(HmBlocks, 1, 1);
	dim3 ThreadsPerBlock(MaxThreadsPerBlock,1,1);

	float *d_arr; hipMalloc(&d_arr, size*size*sizeof(float));
	int *d_perm; hipMalloc(&d_perm, total_threads*size*sizeof(float));
	float *d_result; hipMalloc(&d_result, total_threads*sizeof(float));


	hipMemcpy(d_perm, perm, total_threads*size*sizeof(float), hipMemcpyHostToDevice);
	auto t1 = std::clock();
	hipMemcpy(d_arr, arr, size*size*sizeof(float), hipMemcpyHostToDevice);
		TSP<<<BlocksPerGrid, ThreadsPerBlock>>>(d_perm, d_result, d_arr, size, total_threads);
	hipMemcpy(result, d_result, total_threads*sizeof(float), hipMemcpyDeviceToHost);
	auto t2 = std::clock();

	hipFree(d_arr);
	hipFree(d_perm);
	hipFree(d_result);

	return static_cast<float>(t2-t1)/CLOCKS_PER_SEC;
}


int main(int argc, char *argv[])
{
	int size;
	if (argc == 1)
		size = 2;
	else
		size = std::stoi(argv[1]);

	float *arr = new float[size*size];
	matrix_randomizer(arr, size);

	int total_threads = 1;
	for (int i=2; i<=size; i++)
		total_threads *= i;

	perm = new int[total_threads*size];

	int *a = new int[size];
	for (int i=0; i<size; i++)
		a[i] = i;

	auto t1 = std::clock();
	std::cout<<"permutation started!"<<std::endl;
	heapPermutation(a, size, size);
	std::cout<<"permutation done!"<<std::endl;
	auto t2 = std::clock();
	std::cout<<"permutation time: "<<1000000*static_cast<float>(t2-t1)/CLOCKS_PER_SEC<<" (microsec)"<<std::endl;

	delete a;

	float *result = new float[total_threads];

	auto cpu_time = cpu_call(arr, perm, result, size, total_threads);
	auto gpu_time = gpu_call(arr, perm, result, size, total_threads);
	
		
	std::cout<<"cpu time = "<<1000000*cpu_time<<" (microsec)"<<std::endl;
	std::cout<<"gpu time = "<<1000000*gpu_time<<" (mircosec)"<<std::endl;

	delete perm;
	delete arr;

}	
